
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void whoami(void) {
    int block_id = 
        blockIdx.x + 
        blockIdx.y * gridDim.x +
        blockIdx.z * gridDim.x * gridDim.y;

    int block_offset = 
        block_id *
        blockDim.x * blockDim.y * blockDim.z;

    int thread_offset =
        threadIdx.x + 
        threadIdx.y * blockDim.x + 
        threadIdx.z * blockDim.x * blockDim.y;

    int id = block_offset + thread_offset;


    printf("%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d\n",
        id,
        blockIdx.x, blockIdx.y, blockIdx.z, block_id,
        threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);
    // printf("blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);

}



int main(int argc, char **argv) {
    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 4, t_y = 4, t_z = 4;


    int blocks_per_grid = b_x * b_y * b_z;
    int threads_per_block = t_x * t_y * t_z;

    printf("%d blocks per grid\n", blocks_per_grid);
    printf("%d threads per block\n", threads_per_block);
    printf("%d total threads\n", blocks_per_grid * threads_per_block);

    dim3 blocksPerGrid(b_x, b_y, b_z); // 3D Cube of shape 2*3*4 = 24
    dim3 threadsPerBlock(t_x, t_y, t_z); // 3D Cube of shape 4*4*4 = 64

    // We need this special syntax to input how many blocks there are per grid, and how many threads there are per block
    whoami<<<blocksPerGrid, threadsPerBlock>>>();
    hipDeviceSynchronize();
}