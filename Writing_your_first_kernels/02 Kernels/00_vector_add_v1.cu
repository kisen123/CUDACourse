#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 10000000  // Vector size = 10 million
#define BLOCK_SIZE 256

// Example:
// A = [1, 2, 3, 4, 5]
// B = [6, 7, 8, 9, 10]
// C = A + B = [7, 9, 11, 13, 15]

// CPU vector addition - COMPLETE
void vector_add_cpu(float *a, float *b, float *c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

// CUDA kernel for vector addition
__global__ void vector_add_gpu(float *a, float *b, float *c, int n) {
    
}

// Initialize vector with random float values - COMPLETE
void init_vector(float* vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX; // RAND_MAX is contained in the stdlib.h file. vec will be between 0 and 1
    }
}

// Function to measure execution time
double get_time() {
    
}


int main() {


    float *h_a, *h_b, *h_c_cpu, *h_c_gpu;
    float *d_a, *d_b, *d_c;

    size_t size = N * sizeof(float);

    // We need to allocate the host memory
    // We also need to cast the result of malloc(size)
    // to be a pointer to a float type. That is why it says (float*)
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c_cpu = (float*)malloc(size);
    h_c_gpu = (float*)malloc(size);

    // Initialize all the vectors
    srand(time(NULL));
    init_vector(h_a, N);
    init_vector(h_b, N);



    return 0;
}